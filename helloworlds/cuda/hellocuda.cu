#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10 * 1000
// * 1000

__global__ void vector_add(float *res, float *a, float *b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        res[i] = a[i] + b[i];
}

int main()
{
    float *a, *b, *res;
    float *d_a, *d_b, *d_res; 

    // Allocate memory
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    res = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipError_t err;
    err = hipMalloc((void**)&d_a, sizeof(float) * N);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    err = hipMalloc((void**)&d_b, sizeof(float) * N);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    err = hipMalloc((void**)&d_res, sizeof(float) * N);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    err = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    err = hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    vector_add<<<ceil(N/256.0),256>>>(d_res, d_a, d_b, N);
    
    err = hipMemcpy(res, d_res, sizeof(float) * N, hipMemcpyDeviceToHost);
    if(err != hipSuccess) 
    {
        printf("Error %s:%d: %s\n",__FILE__, __LINE__, hipGetErrorString(err));
        exit(-1);
    }

    for(int i = 0; i < N; i++){
        if(fabs(res[i] - a[i] - b[i]) > 0.00001) {
            printf("ERROR at %i: fabs(res[%i] (%f) - a[%i](%f) - b[%i](%f)) = %f\n", i, i, res[i], i, a[i], i, b[i], fabs(res[i] - a[i] - b[i]));
            exit(-1);
        }
    }
    printf("OK\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(res);
}
